#include <compressors/compactionv2/compaction-compressor.cuh>
#include <compressors/compactionv2/compaction-defines.cuh>
#include <compressors/compactionv2/compaction-encode.cuh>
#include <compressors/compactionv2/compaction-transpose.cuh>
#include <compressors/shared.cuh>
#include <thread>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

namespace gtsst::compressors {
    CompressionConfiguration CompactionV2Compressor::configure_compression(const size_t buf_size) {
        return CompressionConfiguration{.input_buffer_size = buf_size,
                                        .compression_buffer_size = buf_size,
                                        .temp_buffer_size = buf_size * 3,
                                        .min_alignment_input = compactionv2::WORD_ALIGNMENT,
                                        .min_alignment_output = compactionv2::WORD_ALIGNMENT,
                                        .min_alignment_temp = compactionv2::TMP_WORD_ALIGNMENT,
                                        .must_pad_alignment = true,
                                        .block_size = compactionv2::BLOCK_SIZE,
                                        .table_range = compactionv2::BLOCK_SIZE * compactionv2::SUPER_BLOCK_SIZE,
                                        .must_pad_block = true,

                                        .escape_symbol = fsst::Symbol::escape,
                                        .padding_symbol = fsst::Symbol::ignore,
                                        .padding_enabled = true,

                                        .device_buffers = true};
    }

    GTSSTStatus CompactionV2Compressor::validate_compression_buffers(const uint8_t* src, uint8_t* dst, uint8_t* tmp,
                                                                     CompressionConfiguration& config) {
        if (config.input_buffer_size > compactionv2::BLOCK_SIZE * (size_t)0xFFFFFFFF) {
            return gtsstErrorTooBig;
        }

        if (config.block_size != compactionv2::BLOCK_SIZE) {
            return gtsstErrorBadBlockSize;
        }

        if (config.min_alignment_input != compactionv2::WORD_ALIGNMENT ||
            config.min_alignment_output != compactionv2::WORD_ALIGNMENT ||
            config.min_alignment_temp != compactionv2::TMP_WORD_ALIGNMENT) {
            return gtsstErrorBadBlockSize;
        }

        if ((uintptr_t)src % compactionv2::WORD_ALIGNMENT != 0 || (uintptr_t)dst % compactionv2::WORD_ALIGNMENT != 0 ||
            (uintptr_t)tmp % compactionv2::TMP_WORD_ALIGNMENT != 0) {
            return gtsstErrorAlignment;
        }

        if (config.input_buffer_size % compactionv2::BLOCK_SIZE != 0 ||
            config.temp_buffer_size % compactionv2::TMP_OUT_BLOCK_SIZE != 0) {
            return gtsstErrorBlockAlignment;
        }

        if (config.input_buffer_size % compactionv2::WORD_ALIGNMENT != 0 ||
            config.temp_buffer_size % compactionv2::TMP_WORD_ALIGNMENT != 0) {
            return gtsstErrorWordAlignment;
        }

        return gtsstSuccess;
    }

    GTSSTStatus CompactionV2Compressor::compress(const uint8_t* src, uint8_t* dst, const uint8_t* sample_src,
                                                 uint8_t* tmp, CompressionConfiguration& config, size_t* out_size,
                                                 CompressionStatistics& stats) {
        if (const GTSSTStatus buffer_validation = validate_compression_buffers(src, dst, tmp, config);
            buffer_validation != gtsstSuccess) {
            return buffer_validation;
        }

        if (config.input_buffer_size == 0) {
            return gtsstSuccess;
        }

        // TODO: remove this assertion
        assert(!data_contains(sample_src, 254, config.input_buffer_size));
        assert(!data_contains(sample_src, 255, config.input_buffer_size));

        // Some bookkeeping
        const uint64_t number_of_blocks = config.input_buffer_size / compactionv2::BLOCK_SIZE;
        const uint64_t working_block_size = number_of_blocks * compactionv2::TMP_OUT_BLOCK_SIZE;
        const uint64_t number_of_tables = (number_of_blocks - 1) / compactionv2::SUPER_BLOCK_SIZE + 1;
        const uint64_t metadata_mem_size = sizeof(compactionv2::GCompactionMetadata) * number_of_tables;
        const uint64_t block_headers_mem_size = sizeof(BlockHeader) * number_of_blocks;

        compactionv2::GCompactionMetadata* metadata_host;
        GBaseHeader* table_headers_host;
        BlockHeader* block_headers_host;
        safeCUDACall(hipHostMalloc(&metadata_host, metadata_mem_size));
        safeCUDACall(hipHostMalloc(&table_headers_host, sizeof(GBaseHeader) * number_of_tables));
        safeCUDACall(hipHostMalloc(&block_headers_host, block_headers_mem_size));

        // Some CUDA bookkeeping
        compactionv2::GCompactionMetadata* metadata_gpu;
        BlockHeader* block_headers_gpu;

        // Allocate some CUDA buffers
        safeCUDACall(hipMalloc(&metadata_gpu, metadata_mem_size));
        safeCUDACall(hipMalloc(&block_headers_gpu, block_headers_mem_size));

        // Set temp_dst to all ignores, so all unused data is filtered in a later stage
        safeCUDACall(hipMemsetAsync(tmp, 254, working_block_size));

        // Start transpose while table is being generated
        compactionv2::shared_transpose<uint64_t, compactionv2::BLOCK_SIZE, compactionv2::n_words_per_tile,
                                       compactionv2::THREAD_COUNT><<<number_of_blocks, 32>>>(src, dst);

        // Phase 1: Symbol generation (CPU for now)
        const auto symbol_start = std::chrono::high_resolution_clock::now();

        std::vector<std::thread> threads;
        threads.reserve(number_of_tables);
        for (uint32_t i = 0; i < number_of_tables; i++) {
            threads.emplace_back(gpu_create_metadata<symbols::SmallSymbolMatchTableData>, i,
                                 compactionv2::BLOCK_SIZE * compactionv2::SUPER_BLOCK_SIZE, metadata_host,
                                 table_headers_host, sample_src, config.input_buffer_size);
        }
        for (std::thread& t : threads) {
            t.join();
        }

        // Phase 2: Precomputation
        const auto precomputation_start = std::chrono::high_resolution_clock::now();
        // Copy metadata to GPU memory
        safeCUDACall(hipMemcpyAsync(metadata_gpu, metadata_host, metadata_mem_size, hipMemcpyHostToDevice));

        // compactionv2::shared_transpose<<<number_of_blocks, 32>>>(src, dst, compactionv2::BLOCK_SIZE);
        safeCUDACall(hipPeekAtLastError());
        safeCUDACall(hipDeviceSynchronize());

        // Phase 3: Encoding (GPU)
        const auto encoding_start = std::chrono::high_resolution_clock::now();

        // Run all blocks
        compactionv2::gpu_compaction<<<number_of_blocks, compactionv2::THREAD_COUNT>>>(metadata_gpu, block_headers_gpu,
            dst, tmp);
        safeCUDACall(hipPeekAtLastError());
        safeCUDACall(hipDeviceSynchronize());

        // Phase 4: Postprocessing (Partial CPU for now)
        const auto post_start = std::chrono::high_resolution_clock::now();

        // Copy comp headers & temp_dst to CPU
        safeCUDACall(hipMemcpy(block_headers_host, block_headers_gpu, block_headers_mem_size, hipMemcpyDeviceToHost));

        // Gather total output size
        uint64_t total_data_size = 0;
        for (uint32_t block_id = 0; block_id < number_of_blocks; block_id++) {
            total_data_size += block_headers_host[block_id].compressed_size;
        }

        // Calculate header positions
        FileHeader file_header{
            .compressed_size = total_data_size + block_headers_mem_size + sizeof(FileHeader),
            .uncompressed_size = config.input_buffer_size, // TODO: would have to change this to support input padding
            .num_tables = (uint32_t)number_of_tables,
            .table_size = 0,
            .num_blocks = (uint32_t)number_of_blocks,
        };
        size_t header_size = sizeof(FileHeader);

        // Copy tables
        for (int table_id = 0; table_id < number_of_tables; table_id++) {
            safeCUDACall(hipMemcpyAsync(dst + header_size, &table_headers_host[table_id],
                metadata_host[table_id].header_offset, hipMemcpyHostToDevice));

            header_size += metadata_host[table_id].header_offset;
            file_header.table_size += metadata_host[table_id].header_offset;
        }

        // Copy block headers
        safeCUDACall(
            hipMemcpyAsync(dst + header_size, block_headers_host, block_headers_mem_size, hipMemcpyHostToDevice));
        header_size += block_headers_mem_size;

        // Copy file header
        file_header.compressed_size += file_header.table_size;
        safeCUDACall(hipMemcpyAsync(dst, &file_header, sizeof(FileHeader), hipMemcpyHostToDevice));

        // Then do stream compaction on the actual data
        const thrust::device_ptr<uint8_t> thrust_gpu_in = thrust::device_pointer_cast(tmp);
        const thrust::device_ptr<uint8_t> thrust_gpu_out = thrust::device_pointer_cast(dst + header_size);
        const thrust::device_ptr<uint8_t> thrust_new_end =
            copy_if(thrust::device, thrust_gpu_in, thrust_gpu_in + working_block_size, thrust_gpu_out, is_not_ignore());
        const size_t thrust_out_size = thrust_new_end - thrust_gpu_out;
        const size_t out = thrust_out_size + header_size;

        // Finally, free buffers
        safeCUDACall(hipHostFree(metadata_host));
        safeCUDACall(hipHostFree(table_headers_host));
        safeCUDACall(hipHostFree(block_headers_host));

        // And free cuda buffers
        safeCUDACall(hipFree(metadata_gpu));
        safeCUDACall(hipFree(block_headers_gpu));

        // Check and update output size
        assert(file_header.compressed_size - sizeof(FileHeader) - file_header.table_size -
            block_headers_mem_size ==
            total_data_size);
        assert(thrust_out_size == total_data_size);
        *out_size = out;

        // Update statistics
        stats.table_generation =
            std::chrono::duration_cast<std::chrono::microseconds>(precomputation_start - symbol_start);
        stats.precomputation =
            std::chrono::duration_cast<std::chrono::microseconds>(encoding_start - precomputation_start);
        stats.encoding = std::chrono::duration_cast<std::chrono::microseconds>(post_start - encoding_start);
        stats.postprocessing = std::chrono::duration_cast<std::chrono::microseconds>(
            std::chrono::high_resolution_clock::now() - post_start);

        return gtsstSuccess;
    }
} // namespace gtsst::compressors
